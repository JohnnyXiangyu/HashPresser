#include "hip/hip_runtime.h"
#include "Acceleration.cuh"
#include "StringGenerator.h"
#include "SHA256.h"

#include <iostream>

using namespace std;

__global__ void PerCharacter(std::string targetDigest, int targetLength, string* decodeResult, bool* done)
{
	// TODO: move all logic inside this block (calling host function from device function is not allowed)
	StringGenerator gen(targetLength - 1);
	unsigned char head = threadIdx.x; // TODO: figure out what's wrong with thread index

	while (!gen.Depleted() && !(*done))
	{
		string tail = "";
		tail += head;
		tail += gen.Peek();

		string currentDigest = "";
		{
			SHA256 sha;
			sha.update(tail);

			uint8_t* digest = sha.digest();
			currentDigest = SHA256::toString(digest);
			delete[] digest;
		}

		if (currentDigest == targetDigest)
		{
			*done = true;
			*decodeResult = tail;
			return;
		}
		else
		{
			gen.Next();
		}
	}
}
